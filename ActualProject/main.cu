#include "hip/hip_runtime.h"
﻿
#include "defines.h"
#include "Platform.h"
#include "vec3.h"
#include "Image.h"
#include "Blocks.h"
#include "Timer.h"
#include "Ray.h"
#include "Camera.h"
#include "HittableList.h"
#include "HittableObject.h"
#include "HittableSphere.h"
#include "Material.h"
#include "Filesystem.h"


RTX_GLOBAL void renderInit(u32 imageWidth, u32 imageHeight, hiprandState* pRandState) {
    const u32 i{ threadIdx.x + blockIdx.x * blockDim.x };
    const u32 j{ threadIdx.y + blockIdx.y * blockDim.y };
    if ((i >= imageWidth) || (j >= imageHeight)) { return; }
    const u32 index{ j * imageWidth + i };
    hiprand_init(1984, index, 0, &pRandState[index]);
}


RTX_DEVICE vec3 colorRay(const Ray& ray, HittableObject** pWorld, hiprandState* pRandState, i32 recursionDepth) {
    const color white{ 1.f, 1.f, 1.f };
    const color blue{ 0.5f, 0.7f, 1.f };

    Ray currentRay{ ray };
    vec3 currentAttenuation{ 1.f, 1.f, 1.f };

    for (i32 i = 0; i < recursionDepth; i++) {
        HitSpecification hitSpecs;
        if ((*pWorld)->hit(currentRay, { 0.001f, FLT_MAX }, &hitSpecs)) {
            Ray scatteredRay;
            vec3 attenuation;
            if (hitSpecs.pMaterial->scatter(currentRay, hitSpecs, &attenuation, &scatteredRay, pRandState)) {
                currentAttenuation = currentAttenuation * attenuation;
                currentRay = scatteredRay;
            }
            else {
                return vec3{};
            }
        }
        else {
            const vec3 unitRayDirection{ vec3::normalize(currentRay.direction) };
            const f32 t{ 0.5f * (unitRayDirection.y + 1.f) };
            const vec3 heavenColor{ (1.f - t) * white + t * blue };
            return currentAttenuation * heavenColor;
        }
    }

    return { 0.f, 0.f, 0.f };
}


RTX_DEVICE f32 clamp(f32 x, f32 min, f32 max) {
    if (x < min) {
        return min;
    }
    if (x > max) {
        return max;
    }
    return x;
}


RTX_DEVICE color applyPostProcessing(color pixel, i32 samplesPerPixel) {
    const f32 scale{ 1.f / (f32)samplesPerPixel };
    color sampledPixel{ pixel * scale };
    if constexpr (ENABLE_GAMMA_CORRECTION) {
        sampledPixel = vec3::square(sampledPixel);
    }
    return {
        255.999f * clamp(sampledPixel.r, 0.f, 0.999f),
        255.999f * clamp(sampledPixel.g, 0.f, 0.999f),
        255.999f * clamp(sampledPixel.b, 0.f, 0.999f)
    };
}


RTX_GLOBAL void render(color* pPixels, u32 imageWidth, u32 imageHeight, u32 samples, i32 recursionDepth,
                       Camera** pCamera, HittableObject** pWorld, hiprandState* pRandState) {
    const u32 i{ threadIdx.x + blockIdx.x * blockDim.x };
    const u32 j{ threadIdx.y + blockIdx.y * blockDim.y };
    if ((i >= imageWidth) || (j >= imageHeight)) { return; }
    const u32 index{ j * imageWidth + i };

    hiprandState localRandState = pRandState[index];
    color localPixel{ 0.f, 0.f, 0.f };

    for (u32 s = 0; s < samples; s++) {
        const f32 u{ ((f32)i + hiprand_uniform(&localRandState)) / (f32)imageWidth };
        const f32 v{ ((f32)j + hiprand_uniform(&localRandState)) / (f32)imageHeight };
        const Ray ray{ (*pCamera)->origin(), (*pCamera)->calculateRayDirection(u, v) };
        localPixel = localPixel + colorRay(ray, pWorld, &localRandState, recursionDepth);
    }
    pRandState[index] = localRandState;

    pPixels[index] = applyPostProcessing(localPixel, samples);
}


RTX_GLOBAL void renderClose(Camera** pCamera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *pCamera;
    }
}


RTX_GLOBAL void worldCreate(HittableObject** pList, HittableObject** pWorld, Camera** pCamera, f32 aspectRatio, u32 listCount) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(pList + 0) = new HittableSphere{ point3{  0.0f,    0.0f,  -1.f}, radius{  0.5f }, new Metal{      color{ 0.8f, 0.8f, 0.8f } } };
        *(pList + 1) = new HittableSphere{ point3{  1.5f,    0.0f,  -1.f}, radius{  0.5f }, new Lambertian{ color{ 0.7f, 0.3f, 0.3f } } };
        *(pList + 2) = new HittableSphere{ point3{ -1.5f,    0.0f,  -2.f}, radius{  0.5f }, new Lambertian{ color{ 0.2f, 0.3f, 0.7f } } };
        *(pList + 3) = new HittableSphere{ point3{ -1.0f,   -0.2f,  -1.f}, radius{  0.3f }, new Metal{      color{ 0.8f, 0.6f, 0.2f } } };
        *(pList + 4) = new HittableSphere{ point3{  0.0f, -100.5f,  -1.f}, radius{ 100.f }, new Lambertian{ color{ 0.8f, 0.8f, 0.f  } } };
        *pWorld = new HittableList(pList, listCount);

        CameraSpecification camSpecs;
        camSpecs.height = 2.f;
        camSpecs.width = camSpecs.height * aspectRatio;
        camSpecs.focalLength = 1.f;
        camSpecs.origin = point3{ 0.f, 0.f, 0.f };

        *pCamera = new Camera();
        (*pCamera)->initialize(camSpecs);
    }
}


RTX_GLOBAL void worldFree(HittableObject** pList, HittableObject** pWorld, u32 listCount) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (u32 i = 0; i < listCount; i++) {
            (*(pList + i))->deleteMaterial();
            delete *(pList + i);
        }
        delete *pWorld;
    }
}


void printCrucialInfoAboutRendering(Image* pImage, Blocks* pBlocks) {
    std::cerr << "Rendering a " << pImage->getWidth() << "x" << pImage->getHeight() << " image "
              << "with " << pImage->getSamples() << " samples per pixel in " << pBlocks->getWidth() << "x"
              << pBlocks->getHeight() << " blocks.\n";
}


auto main() -> i32 {
    FilesystemSpecification filesystemSpecs;
    Filesystem filesystem;
    filesystem.load("resources/default.json", &filesystemSpecs);

    return;

    ImageSpecification imageSpecs{};
    imageSpecs.width = 720;
    imageSpecs.height = 405;
    imageSpecs.samplesPerPixel = 20;
    imageSpecs.recursionDepth = 50;
    
    Image image{};
    image.initialize(imageSpecs);

    BlocksSpecification blockSpecs{};
    blockSpecs.x = 8;
    blockSpecs.y = 8;

    Blocks blocks{};
    blocks.initialize(blockSpecs, &image);

    printCrucialInfoAboutRendering(&image, &blocks);

    hiprandState* pRandState;
    CUDA_CHECK( hipMalloc((void**)&pRandState, image.getCount() * sizeof(hiprandState)));

    const u32 listCount{ 5 };
    HittableObject** pList;
    CUDA_CHECK( hipMalloc((void**)&pList, listCount * sizeof(HittableObject*)) );
    HittableObject** pWorld;
    CUDA_CHECK( hipMalloc((void**)&pWorld, 1 * sizeof(HittableObject*)) );
    Camera** pCamera;
    CUDA_CHECK( hipMalloc((void**)&pCamera, sizeof(Camera*)) );

    RTX_CALL_KERNEL_AND_VALIDATE( worldCreate<<<1, 1>>>(pList, pWorld, pCamera, image.getAspectRatio(), listCount) );
    RTX_CALL_KERNEL_AND_VALIDATE( 
        renderInit<<<blocks.getBlocks(), blocks.getThreads()>>>(image.getWidth(), image.getHeight(), pRandState)
    );

    Timer<TimerType::MILISECONDS> timer;
    timer.start();

    RTX_CALL_KERNEL_AND_VALIDATE(
        render<<<blocks.getBlocks(), blocks.getThreads()>>>(image.getPixels(),
                                                            image.getWidth(),
                                                            image.getHeight(),
                                                            image.getSamples(),
                                                            image.getDepth(),
                                                            pCamera,
                                                            pWorld,
                                                            pRandState)
    );

    timer.stop();

    RTX_CALL_KERNEL_AND_VALIDATE( renderClose<<<1, 1>>>(pCamera) );
    RTX_CALL_KERNEL_AND_VALIDATE( worldFree<<<1, 1>>>(pList, pWorld, listCount) );

    CUDA_CHECK( hipFree(pRandState) );
    CUDA_CHECK( hipFree(pCamera) );
    CUDA_CHECK( hipFree(pList) );
    CUDA_CHECK( hipFree(pWorld) );

    writeImageToFile("output_image.ppm", &image);
    image.free();

    return 0;
}
