#include "hip/hip_runtime.h"
﻿
#include "defines.h"
#include "Platform.h"
#include "vec3.h"
#include "Image.h"
#include "Blocks.h"
#include "Timer.h"
#include "Ray.h"
#include "Camera.h"
#include "HittableList.h"
#include "HittableObject.h"
#include "HittableSphere.h"
#include "Material.h"
#include "Filesystem.h"


RTX_GLOBAL void renderInit(u32 imageWidth, u32 imageHeight, hiprandState* pRandState) {
    const u32 i{ threadIdx.x + blockIdx.x * blockDim.x };
    const u32 j{ threadIdx.y + blockIdx.y * blockDim.y };
    if ((i >= imageWidth) || (j >= imageHeight)) { return; }
    const u32 index{ j * imageWidth + i };
    hiprand_init(1984, index, 0, &pRandState[index]);
}


RTX_DEVICE vec3 colorRay(const Ray& ray, HittableObject** pWorld, hiprandState* pRandState, i32 recursionDepth) {
    const color white{ 1.f, 1.f, 1.f };
    const color blue{ 0.5f, 0.7f, 1.f };

    Ray currentRay{ ray };
    vec3 currentAttenuation{ 1.f, 1.f, 1.f };

    for (i32 i = 0; i < recursionDepth; i++) {
        HitSpecification hitSpecs;
        if ((*pWorld)->hit(currentRay, { 0.001f, FLT_MAX }, &hitSpecs)) {
            Ray scatteredRay;
            vec3 attenuation;
            if (hitSpecs.pMaterial->scatter(currentRay, hitSpecs, &attenuation, &scatteredRay, pRandState)) {
                currentAttenuation = currentAttenuation * attenuation;
                currentRay = scatteredRay;
            }
            else {
                return vec3{};
            }
        }
        else {
            const vec3 unitRayDirection{ vec3::normalize(currentRay.direction) };
            const f32 t{ 0.5f * (unitRayDirection.y + 1.f) };
            const vec3 heavenColor{ (1.f - t) * white + t * blue };
            return currentAttenuation * heavenColor;
        }
    }

    return { 0.f, 0.f, 0.f };
}


RTX_DEVICE f32 clamp(f32 x, f32 min, f32 max) {
    if (x < min) {
        return min;
    }
    if (x > max) {
        return max;
    }
    return x;
}


RTX_DEVICE color applyPostProcessing(color pixel, i32 samplesPerPixel) {
    const f32 scale{ 1.f / (f32)samplesPerPixel };
    color sampledPixel{ pixel * scale };
    if constexpr (ENABLE_GAMMA_CORRECTION) {
        sampledPixel = vec3::square(sampledPixel);
    }
    return {
        255.999f * clamp(sampledPixel.r, 0.f, 0.999f),
        255.999f * clamp(sampledPixel.g, 0.f, 0.999f),
        255.999f * clamp(sampledPixel.b, 0.f, 0.999f)
    };
}


RTX_GLOBAL void render(color* pPixels, u32 imageWidth, u32 imageHeight, u32 samples, i32 recursionDepth,
                       Camera** pCamera, HittableObject** pWorld, hiprandState* pRandState) {
    const u32 i{ threadIdx.x + blockIdx.x * blockDim.x };
    const u32 j{ threadIdx.y + blockIdx.y * blockDim.y };
    if ((i >= imageWidth) || (j >= imageHeight)) { return; }
    const u32 index{ j * imageWidth + i };

    hiprandState localRandState = pRandState[index];
    color localPixel{ 0.f, 0.f, 0.f };

    for (u32 s = 0; s < samples; s++) {
        const f32 u{ ((f32)i + hiprand_uniform(&localRandState)) / (f32)imageWidth };
        const f32 v{ ((f32)j + hiprand_uniform(&localRandState)) / (f32)imageHeight };
        const Ray ray{ (*pCamera)->origin(), (*pCamera)->calculateRayDirection(u, v) };
        localPixel = localPixel + colorRay(ray, pWorld, &localRandState, recursionDepth);
    }
    pRandState[index] = localRandState;

    pPixels[index] = applyPostProcessing(localPixel, samples);
}


RTX_GLOBAL void renderClose(Camera** pCamera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *pCamera;
    }
}


RTX_GLOBAL void worldCreate(HittableObject** pList, HittableObject** pWorld, Camera** pCamera,
                            f32 aspectRatio, u32 listCount,
                            f32* positions, f32* colors, i32* materials, f32* radius) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (u32 i = 0; i < listCount; i++) {
            Material* pMaterial = materials[i] == 0 ? (Material*)new Metal{} : (Material*)new Lambertian{};
            pMaterial->setAlbedo(colors[i * 3 + 0], colors[i * 3 + 1], colors[i * 3 + 2]);
            *(pList + i) = new HittableSphere{
                point3{ positions[i * 3 + 0], positions[i * 3 + 1], positions[i * 3 + 2] },
                f32{ radius[i] },
                pMaterial
            };
        }
        *pWorld = new HittableList(pList, listCount);

        CameraSpecification camSpecs;
        camSpecs.height = 2.f;
        camSpecs.width = camSpecs.height * aspectRatio;
        camSpecs.focalLength = 1.f;
        camSpecs.origin = point3{ 0.f, 0.f, 0.f };

        *pCamera = new Camera();
        (*pCamera)->initialize(camSpecs);
    }
}


RTX_GLOBAL void worldFree(HittableObject** pList, HittableObject** pWorld, u32 listCount) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (u32 i = 0; i < listCount; i++) {
            (*(pList + i))->deleteMaterial();
            delete *(pList + i);
        }
        delete *pWorld;
    }
}


void printCrucialInfoAboutRendering(Image* pImage, Blocks* pBlocks) {
    std::cerr << "Rendering a " << pImage->getWidth() << "x" << pImage->getHeight() << " image "
              << "with " << pImage->getSamples() << " samples per pixel in " << pBlocks->getWidth() << "x"
              << pBlocks->getHeight() << " blocks.\n";
}


auto main() -> i32 {
    FilesystemSpecification filesystemSpecs;
    Filesystem filesystem;
    filesystem.load("resources/second.json", &filesystemSpecs);

    ImageSpecification imageSpecs{};
    imageSpecs.width = 720;
    imageSpecs.height = 405;
    imageSpecs.samplesPerPixel = 20;
    imageSpecs.recursionDepth = 50;
    
    Image image{};
    image.initialize(imageSpecs);

    BlocksSpecification blockSpecs{};
    blockSpecs.x = 8;
    blockSpecs.y = 8;

    Blocks blocks{};
    blocks.initialize(blockSpecs, &image);

    printCrucialInfoAboutRendering(&image, &blocks);

    hiprandState* pRandState;
    CUDA_CHECK( hipMalloc((void**)&pRandState, image.getCount() * sizeof(hiprandState)));

    u32 itemsCount{ (u32)filesystemSpecs.materials.size() };
    HittableObject** pList;
    CUDA_CHECK( hipMalloc((void**)&pList, itemsCount * sizeof(HittableObject*)) );
    HittableObject** pWorld;
    CUDA_CHECK( hipMalloc((void**)&pWorld, 1 * sizeof(HittableObject*)) );
    Camera** pCamera;
    CUDA_CHECK( hipMalloc((void**)&pCamera, sizeof(Camera*)) );

    f32* positionsGPU;
    hipMalloc(&positionsGPU, filesystemSpecs.positions.size() * sizeof(decltype(filesystemSpecs.positions[0])));
    hipMemcpy(positionsGPU, filesystemSpecs.positions.data(), filesystemSpecs.positions.size() * sizeof(decltype(filesystemSpecs.positions[0])), hipMemcpyHostToDevice);

    f32* colorsGPU;
    hipMalloc(&colorsGPU, filesystemSpecs.colors.size() * sizeof(decltype(filesystemSpecs.colors[0])));
    hipMemcpy(colorsGPU, filesystemSpecs.colors.data(), filesystemSpecs.colors.size() * sizeof(decltype(filesystemSpecs.colors[0])), hipMemcpyHostToDevice);

    i32* materialsGPU;
    hipMalloc(&materialsGPU, filesystemSpecs.materials.size() * sizeof(decltype(filesystemSpecs.materials[0])));
    hipMemcpy(materialsGPU, filesystemSpecs.materials.data(), filesystemSpecs.materials.size() * sizeof(decltype(filesystemSpecs.materials[0])), hipMemcpyHostToDevice);

    f32* radiusGPU;
    hipMalloc(&radiusGPU, filesystemSpecs.radius.size() * sizeof(decltype(filesystemSpecs.radius[0])));
    hipMemcpy(radiusGPU, filesystemSpecs.radius.data(), filesystemSpecs.radius.size() * sizeof(decltype(filesystemSpecs.radius[0])), hipMemcpyHostToDevice);

    RTX_CALL_KERNEL_AND_VALIDATE(
        worldCreate<<<1, 1>>>(pList, pWorld, pCamera, image.getAspectRatio(), itemsCount,
                              positionsGPU, colorsGPU,
                              materialsGPU, radiusGPU) );
    RTX_CALL_KERNEL_AND_VALIDATE(
        renderInit<<<blocks.getBlocks(), blocks.getThreads()>>>(image.getWidth(), image.getHeight(), pRandState)
    );

    Timer<TimerType::MILISECONDS> timer;
    timer.start();

    RTX_CALL_KERNEL_AND_VALIDATE(
        render<<<blocks.getBlocks(), blocks.getThreads()>>>(image.getPixels(),
                                                            image.getWidth(),
                                                            image.getHeight(),
                                                            image.getSamples(),
                                                            image.getDepth(),
                                                            pCamera,
                                                            pWorld,
                                                            pRandState)
    );

    timer.stop();

    RTX_CALL_KERNEL_AND_VALIDATE( renderClose<<<1, 1>>>(pCamera) );
    RTX_CALL_KERNEL_AND_VALIDATE( worldFree<<<1, 1>>>(pList, pWorld, itemsCount) );

    CUDA_CHECK( hipFree(pRandState) );
    CUDA_CHECK( hipFree(pCamera) );
    CUDA_CHECK( hipFree(pList) );
    CUDA_CHECK( hipFree(pWorld) );

    writeImageToFile("output_image.ppm", &image);
    image.free();

    hipFree(positionsGPU);
    hipFree(colorsGPU);
    hipFree(materialsGPU);
    hipFree(radiusGPU);

    return 0;
}
