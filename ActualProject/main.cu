#include "hip/hip_runtime.h"
﻿
#include "defines.h"
#include "vec3.h"


// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(vec3* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    fb[pixel_index] = 255.99f * vec3(float(i) / max_x, float(j) / max_y, 0.2f);
}


static void writePixelToFile(std::ostream& out, vec3 pixel) {
    out << (i32)(pixel.r) << ' '
        << (i32)(pixel.g) << ' '
        << (i32)(pixel.b) << '\n';
}


void writeImageToFile(const char* outputPath, i32 width, i32 height, i32 pixelsCount,
                      vec3* pImage) {
    std::ofstream file;
    file.open(outputPath);
    file << "P3\n" << width << ' ' << height << "\n255\n";

    for (i32 i = 0; i < pixelsCount; i++) {
        writePixelToFile(file, pImage[i]);
    }

    file.close();
}


auto main() -> i32 {
    int nx = 720;
    int ny = 405;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3* fb{ nullptr };
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    writeImageToFile("output_image.ppm", nx, ny, num_pixels, fb);

    checkCudaErrors(hipFree(fb));
}
